#include "hip/hip_runtime.h"
#include "../include/gates_cuda.cuh"
#include <complex>
#include <hip/hip_complex.h> // Use CUDA's complex number header
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <vector>

// Helper macro to check for CUDA errors
#define CUDA_CHECK(err)                                                        \
  {                                                                            \
    if (err != hipSuccess) {                                                  \
      throw std::runtime_error(std::string("CUDA Error: ") +                   \
                               hipGetErrorString(err));                       \
    }                                                                          \
  }

// --- CUDA Kernels (Device Code) ---
// Note: All kernels now use hipDoubleComplex instead of std::complex<double>

__global__ void h_gate_kernel(hipDoubleComplex *amplitudes, int num_qubits,
                              int target_qubit) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  size_t num_amplitudes = 1ULL << num_qubits;

  if (i < num_amplitudes) {
    if (!((i >> target_qubit) & 1)) {
      size_t j = i | (1ULL << target_qubit);

      hipDoubleComplex c_i = amplitudes[i];
      hipDoubleComplex c_j = amplitudes[j];

      const double inv_sqrt2 = 1.0 / 1.4142135623730951;

      // Use hipCadd and hipCsub for complex arithmetic if direct operators fail,
      // but modern CUDA often overloads them correctly.
      amplitudes[i] = make_hipDoubleComplex(hipCreal(c_i) + hipCreal(c_j),
                                           hipCimag(c_i) + hipCimag(c_j));
      amplitudes[i] = hipCmul(amplitudes[i], make_hipDoubleComplex(inv_sqrt2, 0));

      amplitudes[j] = make_hipDoubleComplex(hipCreal(c_i) - hipCreal(c_j),
                                           hipCimag(c_i) - hipCimag(c_j));
      amplitudes[j] = hipCmul(amplitudes[j], make_hipDoubleComplex(inv_sqrt2, 0));
    }
  }
}

__global__ void x_gate_kernel(hipDoubleComplex *amplitudes, int num_qubits,
                              int target_qubit) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  size_t num_amplitudes = 1ULL << num_qubits;

  if (i < num_amplitudes) {
    if (!((i >> target_qubit) & 1)) {
      size_t j = i | (1ULL << target_qubit);
      // Replace std::swap with a manual swap
      hipDoubleComplex temp = amplitudes[i];
      amplitudes[i] = amplitudes[j];
      amplitudes[j] = temp;
    }
  }
}

__global__ void y_gate_kernel(hipDoubleComplex *amplitudes, int num_qubits,
                              int target_qubit) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  size_t num_amplitudes = 1ULL << num_qubits;
  const hipDoubleComplex imag_i = make_hipDoubleComplex(0.0, 1.0);

  if (i < num_amplitudes) {
    if (!((i >> target_qubit) & 1)) {
      size_t j = i | (1ULL << target_qubit);
      hipDoubleComplex c_i = amplitudes[i];
      hipDoubleComplex c_j = amplitudes[j];
      amplitudes[i] = hipCmul(make_hipDoubleComplex(0.0, -1.0), c_j); // -i * c_j
      amplitudes[j] = hipCmul(imag_i, c_i);                          //  i * c_i
    }
  }
}

__global__ void cnot_gate_kernel(hipDoubleComplex *amplitudes, int num_qubits,
                                 int control_qubit, int target_qubit) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  size_t num_amplitudes = 1ULL << num_qubits;

  if (i < num_amplitudes) {
    if (((i >> control_qubit) & 1) && !((i >> target_qubit) & 1)) {
      size_t j = i | (1ULL << target_qubit);
      hipDoubleComplex temp = amplitudes[i];
      amplitudes[i] = amplitudes[j];
      amplitudes[j] = temp;
    }
  }
}

// --- Host-Callable Wrappers (Host Code) ---
// This is a generic helper function to reduce code duplication
void launch_kernel(QuantumState &state, const void *kernel, int target_qubit,
                   int control_qubit = -1) {
  size_t num_amplitudes = state.amplitudes.size();
  size_t vector_size_bytes = num_amplitudes * sizeof(hipDoubleComplex);
  hipDoubleComplex *d_amplitudes;

  CUDA_CHECK(hipMalloc(&d_amplitudes, vector_size_bytes));
  // Use reinterpret_cast because std::complex and hipDoubleComplex have
  // compatible memory layouts
  CUDA_CHECK(hipMemcpy(d_amplitudes,
                        reinterpret_cast<const void *>(state.amplitudes.data()),
                        vector_size_bytes, hipMemcpyHostToDevice));

  int threads_per_block = 256;
  int blocks_per_grid =
      (num_amplitudes + threads_per_block - 1) / threads_per_block;

  // Launch the correct kernel based on the function pointer
  if (kernel == (const void *)h_gate_kernel) {
    h_gate_kernel<<<blocks_per_grid, threads_per_block>>>(
        d_amplitudes, state.num_qubits, target_qubit);
  } else if (kernel == (const void *)x_gate_kernel) {
    x_gate_kernel<<<blocks_per_grid, threads_per_block>>>(
        d_amplitudes, state.num_qubits, target_qubit);
  } else if (kernel == (const void *)y_gate_kernel) {
    y_gate_kernel<<<blocks_per_grid, threads_per_block>>>(
        d_amplitudes, state.num_qubits, target_qubit);
  } else if (kernel == (const void *)cnot_gate_kernel) {
    cnot_gate_kernel<<<blocks_per_grid, threads_per_block>>>(
        d_amplitudes, state.num_qubits, control_qubit, target_qubit);
  }

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(reinterpret_cast<void *>(state.amplitudes.data()),
                        d_amplitudes, vector_size_bytes,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_amplitudes));
}

void apply_H_gate_cuda(QuantumState &state, int target_qubit) {
  launch_kernel(state, (const void *)h_gate_kernel, target_qubit);
}

void apply_X_gate_cuda(QuantumState &state, int target_qubit) {
  launch_kernel(state, (const void *)x_gate_kernel, target_qubit);
}

void apply_Y_gate_cuda(QuantumState &state, int target_qubit) {
  launch_kernel(state, (const void *)y_gate_kernel, target_qubit);
}

void apply_CNOT_gate_cuda(QuantumState &state, int control_qubit,
                          int target_qubit) {
  launch_kernel(state, (const void *)cnot_gate_kernel, target_qubit,
                control_qubit);
}

void create_ghz_state_cuda(QuantumState &state) {
  if (state.num_qubits < 3) {
    throw std::invalid_argument(
        "GHZ state creation requires at least 3 qubits.");
  }
  std::cout << "\n--- Building GHZ State on GPU ---" << std::endl;
  apply_H_gate_cuda(state, 0);
  for (int i = 1; i < state.num_qubits; ++i) {
    apply_CNOT_gate_cuda(state, 0, i);
  }
  std::cout << "--- GHZ State Creation Complete ---\n" << std::endl;
}
